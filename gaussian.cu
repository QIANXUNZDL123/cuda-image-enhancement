#include "hip/hip_runtime.h"
#include "gaussian.cuh"

constexpr int GAUSSIAN_FILTER_SIZE = 3;

__global__ void k_1D_gaussian_filter(unsigned char *input, int rows, int cols, int mask_dim)
{
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;

	int threadId = tx * cols + ty;
	
	int offset = GAUSSIAN_FILTER_SIZE / 2;
	int newPixelValue = 0;


	if(threadId >= rows * cols){
		return;
	}

	int conv_kernel[GAUSSIAN_FILTER_SIZE][GAUSSIAN_FILTER_SIZE] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};

	for (int r = 0; r < mask_dim; r++)
	{
		for (int c = 0; c < mask_dim; c++)
		{
			if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1))
			{
				newPixelValue += conv_kernel[r][c] * input[(tx - offset + r) * cols + (ty - offset + c)];
			}
			else
			{
				return;
			}
		}
	}

	input[threadId] = static_cast<uchar>(newPixelValue / 16);
}

__global__ void k_1D_gaussian_filter_shared_mem(unsigned char* input, int rows, int cols, int mask_dim)
{
	__shared__ unsigned char cache[32][33];

	int conv_kernel[GAUSSIAN_FILTER_SIZE][GAUSSIAN_FILTER_SIZE] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = tx * cols + ty;

	int offset = GAUSSIAN_FILTER_SIZE / 2;
	int newPixelValue = 0;

	if (threadId >= rows * cols)
	{
		return;
	}

	unsigned int cy = threadIdx.x;
	unsigned int cx = threadIdx.y;

	cache[cx][cy] = input[threadId]; /*load data shared mem*/
	__syncthreads();

	for (int i = 0; i < mask_dim; i++)
	{
		for (int j = 0; j < mask_dim; j++)
		{ /*travel on conv matrix*/
			if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1))
			{
				int x_index = cx - offset + i;
				int y_index = cy - offset + j;
				if (cx == 31 || cx == 0 || cy == 0 || cy == 31)
				{
					newPixelValue += conv_kernel[i][j] * input[(tx - offset + i) * cols + (ty - offset + j)];
				}
				else
				{
					newPixelValue += conv_kernel[i][j] * cache[x_index][y_index];
				}
			}
			else
			{
				return;
			}
		}
	}
	input[threadId] = static_cast<uchar>(newPixelValue / 16);
}

float gaussian_filter_gpu_3D(cv::Mat input_img, cv::Mat *output_img, bool sm)
{
	unsigned char *gpu_input = NULL;

	unsigned char *input = input_img.data;
	unsigned char *output = output_img->data;

	unsigned int cols = input_img.cols;
	unsigned int rows = input_img.rows;
	unsigned int size = rows * cols * sizeof(unsigned char) * 3;

	const uint mask_dim = 3;

	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	CHECK_CUDA_ERROR(hipHostRegister(input,size,0));
	CHECK_CUDA_ERROR(hipMalloc((unsigned char **)&gpu_input, size));
	CHECK_CUDA_ERROR(hipMemcpy(gpu_input, input, size, hipMemcpyHostToDevice));

	if(sm){
		k_3D_gaussian_filter_shared_mem<<<grid, block>>>(gpu_input, rows, cols, mask_dim);
	}
	else{
		k_3D_gaussian_filter<<<grid, block>>>(gpu_input, rows, cols, mask_dim);
	}

	CHECK_CUDA_ERROR(hipMemcpy(output, gpu_input, size, hipMemcpyDeviceToHost));

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);

	hipFree(gpu_input);
	hipDeviceReset();
	return elapsed;
}
__global__ void k_3D_gaussian_filter(unsigned char *input, int rows, int cols, int mask_dim)
{
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty ) * 3;

	int offset = GAUSSIAN_FILTER_SIZE / 2;
	int new_red_val = 0;
	int new_green_val = 0;
	int new_blue_val = 0;

	int conv_kernel[GAUSSIAN_FILTER_SIZE][GAUSSIAN_FILTER_SIZE] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};

	if (threadId >= rows * cols * 3)
	{
		return;
	}
	for (int i = 0; i < mask_dim; i++)
	{
		for (int j = 0; j < mask_dim; j++)
		{
			if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1))
			{
				new_red_val += conv_kernel[i][j] * input[((tx - offset + i) * cols + ty - offset + j) * 3];
				new_green_val += conv_kernel[i][j] * input[((tx - offset + i) * cols + ty - offset + j) * 3 + 1];
				new_blue_val += conv_kernel[i][j] * input[((tx - offset + i) * cols + ty - offset + j) * 3 + 2];
			}
			else
			{
				return;
			}
		}
	}
	input[threadId] = static_cast<uchar>(new_red_val / 16);
	input[threadId + 1] = static_cast<uchar>(new_green_val / 16);
	input[threadId + 2] = static_cast<uchar>(new_blue_val / 16);
}
__global__ void k_3D_gaussian_filter_shared_mem(unsigned char *input, int rows, int cols, int mask_dim)
{
	__shared__ unsigned char cache_red[32][32];
	__shared__ unsigned char cache_green[32][32];
	__shared__ unsigned char cache_blue[32][32];

	int conv_kernel[GAUSSIAN_FILTER_SIZE][GAUSSIAN_FILTER_SIZE] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};

	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = (tx * cols + ty) * 3;

	int offset = GAUSSIAN_FILTER_SIZE / 2;
	int new_red_val = 0;
	int new_green_val = 0;
	int new_blue_val = 0;

	if (threadId >= rows * cols * 3)
	{
		return;
	}

	unsigned int cy = threadIdx.x;
	unsigned int cx = threadIdx.y;

	cache_red[cx][cy] = input[threadId]; /*load data shared mem*/
	cache_green[cx][cy] = input[threadId + 1]; 
	cache_blue[cx][cy] = input[threadId + 2]; 
	__syncthreads();

	for (int i = 0; i < mask_dim; i++)
	{
		for (int j = 0; j < mask_dim; j++)
		{ /*travel on conv matrix*/
			if ((tx > 0 && tx < rows - 1) && (ty > 0 && ty < cols - 1))
			{
				int x_index = cx - offset + i;
				int y_index = cy - offset + j;
				if (cx == 31 || cx == 0 || cy == 0 || cy == 31)
				{
					new_red_val += conv_kernel[i][j] * input[((tx - offset + i) * cols + (ty - offset + j)) * 3];
					new_green_val += conv_kernel[i][j] * input[((tx - offset + i) * cols + (ty - offset + j)) * 3 + 1];
					new_blue_val += conv_kernel[i][j] * input[((tx - offset + i) * cols + (ty - offset + j)) * 3 + 2];
				}
				else
				{
					new_red_val += conv_kernel[i][j] * cache_red[x_index][y_index];
					new_green_val += conv_kernel[i][j] * cache_green[x_index][y_index];
					new_blue_val += conv_kernel[i][j] * cache_blue[x_index][y_index];
				}
			}
			else
			{
				return;
			}
		}
	}

	input[threadId] = static_cast<uchar>(new_red_val / 16);
	input[threadId + 1] = static_cast<uchar>(new_green_val / 16);
	input[threadId + 2] = static_cast<uchar>(new_blue_val / 16);
}

float gaussian_filter_gpu_1D(cv::Mat input_img, cv::Mat *output_img, bool sm)
{
	unsigned char *gpu_input = NULL;

	unsigned char *input = input_img.data;
	unsigned char *output = output_img->data;

	unsigned int cols = input_img.cols;
	unsigned int rows = input_img.rows;
	unsigned int pixels = cols * rows;
	unsigned int size = pixels * sizeof(unsigned char);

	const uint mask_dim = 3;

	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	CHECK_CUDA_ERROR(hipMalloc((unsigned char **)&gpu_input, size));
	CHECK_CUDA_ERROR(hipMemcpy(gpu_input, input, size, hipMemcpyHostToDevice));

	if(sm){
		k_1D_gaussian_filter_shared_mem<<<grid, block>>>(gpu_input, rows, cols, mask_dim);
	}
	else{
		k_1D_gaussian_filter<<<grid, block>>>(gpu_input, rows, cols, mask_dim);
	}

	CHECK_CUDA_ERROR(hipMemcpy(output, gpu_input, size, hipMemcpyDeviceToHost));

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);

	hipFree(gpu_input);
	hipDeviceReset();
	return elapsed;
}

float gaussian_filter_cpu_3D(cv::Mat input_img, cv::Mat *output_img)
{
	int cols = input_img.cols;
	int rows = input_img.rows;

	unsigned char* input = input_img.data;
	unsigned char* output = output_img->data;
	const unsigned short mask_dim = 3;
	
	float kernel[mask_dim][mask_dim] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};
	auto start = std::chrono::steady_clock::now();

	for (int i = 1; i < rows - 1; i++)
	{
		for (int j = 1; j < cols - 1; j++)
		{
			int new_red_val = 0;
			int new_green_val = 0;
			int new_blue_val = 0;
			for (int m = 0; m < mask_dim; m++)
			{
				for (int n = 0; n < mask_dim; n++)
				{
					new_red_val += input[(((i + m - 1) * cols + (j + n - 1))) * 3] * kernel[m][n];
					new_green_val += input[((i + m - 1) * cols + (j + n - 1)) * 3 + 1] * kernel[m][n];
					new_blue_val += input[((i + m - 1) * cols + (j + n - 1) )* 3 + 2] * kernel[m][n];
				}
			}
			output[(i * cols + j) * 3] = new_red_val / 16;
			output[(i * cols + j) * 3 + 1] = new_green_val / 16;
			output[(i * cols + j) * 3 + 2] = new_blue_val / 16;
		}
	}
	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start) / 1000.0f;
	return elapsed.count();
}
float gaussian_filter_cpu_1D(cv::Mat input_img, cv::Mat *output_img)
{
	int cols = input_img.cols;
	int rows = input_img.rows;

	unsigned char* input = input_img.data;
	unsigned char* output = output_img->data;
	const unsigned short mask_dim = 3;
	float kernel[mask_dim][mask_dim] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};
	auto start = std::chrono::steady_clock::now();

	for (int i = 1; i < rows - 1; i++)
	{
		for (int j = 1; j < cols - 1; j++)
		{
			int newPixelValue = 0;
			for (int m = 0; m < mask_dim; m++)
			{
				for (int n = 0; n < mask_dim; n++)
				{
					newPixelValue += input[(i + m - 1) * cols + (j + n - 1)] * kernel[m][n];
				}
			}
			output[i * cols + j] = newPixelValue / 16;
		}
	}
	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start) / 1000.0f;
	return elapsed.count();
}

float gaussian_filter_cpu_parallel_1D(cv::Mat input_img, cv::Mat* output_img)
{
	unsigned char *input = input_img.data;
	unsigned char *output = output_img->data;
	int cols = input_img.cols;
	int rows = input_img.rows;
	const unsigned short mask_dim = 3;
	float kernel[mask_dim][mask_dim] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};

	std::vector<std::thread> threads;
	const int MAX_THREAD_SUPPORT = std::thread::hardware_concurrency();

	int stride = rows / MAX_THREAD_SUPPORT;

	auto start = std::chrono::steady_clock::now();
	for (int i = 0; i < MAX_THREAD_SUPPORT; i++)
	{
		threads.push_back(std::thread([&, i](){
			int range_start = stride * i;
			int range_end = (i == MAX_THREAD_SUPPORT - 1) ? cols : stride * (i + 1);

			for (int r = range_start; r < range_end; r++) { /*row loop*/
				for (int c = 0; c < cols; c++) { /*col loop*/
					if (r > 0 && r < rows - 1 && c > 0 && c < cols - 1) {
						int new_pixel_value = 0;
						for (int mr = 0; mr < mask_dim; mr++) { /*matrix row*/
							for (int mc = 0; mc < mask_dim; mc++) { /*matrix col*/
								int r_index = r + mr - 1;
								int c_index = c + mc - 1;
								new_pixel_value += input[r_index * cols + c_index] * kernel[mr][mc];
							}
						}
						output[r * cols + c] = static_cast<unsigned char>(new_pixel_value / 16);
					}
				}
			} }));
	}
	for (std::thread &th : threads)
	{
		th.join();
	}
	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start) / 1000.0f;
	return elapsed.count();
}
float gaussian_filter_cpu_parallel_3D(cv::Mat input_img, cv::Mat* output_img)
{
	unsigned char *input = input_img.data;
	unsigned char *output = output_img->data;
	int cols = input_img.cols;
	int rows = input_img.rows;
	const unsigned short mask_dim = 3;
	float kernel[mask_dim][mask_dim] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};

	std::vector<std::thread> threads;
	const int MAX_THREAD_SUPPORT = std::thread::hardware_concurrency();

	int stride = rows / MAX_THREAD_SUPPORT;

	auto start = std::chrono::steady_clock::now();
	for (int i = 0; i < MAX_THREAD_SUPPORT; i++)
	{
		threads.push_back(std::thread([&, i]()
									  {
			int range_start = stride * i;
			int range_end = (i == MAX_THREAD_SUPPORT - 1) ? cols : stride * (i + 1);

			for (int r = range_start; r < range_end; r++) { /*row loop*/
				for (int c = 0; c < cols; c++) { /*col loop*/
					if (r > 0 && r < rows - 1 && c > 0 && c < cols - 1) {
						int new_pixel_value_red = 0;
						int new_pixel_value_green = 0;
						int new_pixel_value_blue = 0;
						for (int mr = 0; mr < mask_dim; mr++) { /*matrix row*/
							for (int mc = 0; mc < mask_dim; mc++) { /*matrix col*/
								int r_index = r + mr - 1;
								int c_index = c + mc - 1;
								new_pixel_value_red += input[(r_index * cols + c_index) * 3] * kernel[mr][mc];
								new_pixel_value_green += input[(r_index * cols + c_index) * 3 + 1] * kernel[mr][mc];
								new_pixel_value_blue += input[(r_index * cols + c_index) * 3 + 2] * kernel[mr][mc];

							}
						}
						output[(r * cols + c) * 3] = static_cast<unsigned char>(new_pixel_value_red / 16);
						output[(r * cols + c) * 3 + 1] = static_cast<unsigned char>(new_pixel_value_green / 16);
						output[(r * cols + c) * 3 + 2] = static_cast<unsigned char>(new_pixel_value_blue / 16);
					}
				}
			} }));
	}
	for (std::thread &th : threads)
	{
		th.join();
	}
	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start) / 1000.0f;
	return elapsed.count();
}