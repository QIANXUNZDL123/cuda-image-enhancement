#include "hip/hip_runtime.h"
#include "sobel.cuh"

constexpr int SOBEL_FILTER_SIZE = 3;

__constant__ int filter_x_constant[SOBEL_FILTER_SIZE][SOBEL_FILTER_SIZE] = { 0 };
__constant__ int filter_y_constant[SOBEL_FILTER_SIZE][SOBEL_FILTER_SIZE] = { 0 };

__global__ void sobel_filter(unsigned char* input, unsigned char* output, int cols, int rows, int mask_dim) {
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int threadId = tx * cols + ty;

	int offset = mask_dim / 2;
	int gx = 0;
	int gy = 0;

	if (tx > 0 && tx < rows - 1 && ty > 0 && ty < cols - 1) {
		for (int i = 0; i < mask_dim; i++) {
			for (int j = 0; j < mask_dim; j++) {
				gx += filter_x_constant[i][j] * input[(tx - offset + i) * cols + (ty - offset + j)];
				gy += filter_y_constant[i][j] * input[(tx - offset + i) * cols + (ty - offset + j)];
			}
		}
		output[threadId] = static_cast<unsigned char>(sqrtf(static_cast<float>(gx) * static_cast<float>(gx) + static_cast<float>(gy) * static_cast<float>(gy)));
	}
}
__global__ void shared_sobel_filter(unsigned char* input, unsigned char* output, int cols, int rows, int mask_dim) {
	__shared__ unsigned char cache[34][34];
	int tx = blockIdx.y * blockDim.y + threadIdx.y;
	int ty = blockIdx.x * blockDim.x + threadIdx.x;
	int threadId = tx * cols + ty;

	int offset = mask_dim / 2;
	int gx = 0;
	int gy = 0;

	int cy = threadIdx.x + 1;
	int cx = threadIdx.y + 1;

	cache[cx][cy] = input[tx * cols + ty]; /* Load cache[1::32][1::32]*/
	if (cx == 1) {/*Load left column*/
		cache[0][cy] = input[tx * cols + ty - 1];
		if (cy == 1) {
			cache[0][0] = input[(tx - 1) * cols + ty - 1];
		}
		if (cy == 32) {
			cache[33][0] = input[(tx + 1) * cols + ty - 1];
		}
	}
	if (cx == 32) {/*Load right column*/
		cache[33][cy] = input[tx * cols + ty + 1];
		if (cy == 32) {
			cache[33][33] = input[(tx + 1) * cols + ty + 1];
		}
		if (cy == 1) {
			cache[0][33] = input[(tx - 1) * cols + ty + 1];
		}
	}
	if (cy == 1) {/*Load top row*/
		cache[cx][0] = input[(tx - 1) * cols + ty];
	}
	if (cy == 32) {/*Load bottom row*/
		cache[cx][33] = input[(tx + 1) * cols + ty];
	}
	__syncthreads();

	if (ty > 0 && ty < cols - 1 && tx > 0 && tx < rows - 1) {
		for (int i = 0; i < mask_dim; i++) {
			for (int j = 0; j < mask_dim; j++) {
				gx += filter_x_constant[i][j] * cache[cx - offset + i][cy - offset + j];
				gy += filter_y_constant[i][j] * cache[cx - offset + i][cy - offset + j];
			}
		}
		output[threadId] = static_cast<unsigned char>(sqrtf(static_cast<float>(gx) * static_cast<float>(gx) + static_cast<float>(gy) * static_cast<float>(gy)));
	}
}

float sobel_filter_gpu(cv::Mat* inputImg, cv::Mat outputImg) {
	unsigned char* input = inputImg->data;
	unsigned char* output = outputImg.data;

	unsigned char* gpu_input = NULL;
	unsigned char* gpu_output = NULL;

	unsigned int cols = inputImg->cols;
	unsigned int rows = inputImg->rows;
	unsigned int pixels = cols * rows;
	unsigned int size = pixels * sizeof(unsigned char);

	const unsigned int mask_dim = 3;
	int filter_x[3][3] = { {-1 , 0 , 1} , {-2 , 0 , 2 } , {-1 , 0 , 1} };
	int filter_y[3][3] = { {-1 , -2 , -1} , {0 , 0 , 0 } , {1 , 2 , 1} };

	hipEvent_t beginKernel, endKernel, start, stop;
	hipEventCreate(&beginKernel);
	hipEventCreate(&endKernel);
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(filter_x_constant), filter_x, sizeof(int) * SOBEL_FILTER_SIZE * SOBEL_FILTER_SIZE));
	CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(filter_y_constant), filter_y, sizeof(int) * SOBEL_FILTER_SIZE * SOBEL_FILTER_SIZE));
	CHECK_CUDA_ERROR(hipMalloc((unsigned char**)&gpu_input, size));
	CHECK_CUDA_ERROR(hipMalloc((unsigned char**)&gpu_output, size));
	CHECK_CUDA_ERROR(hipMemcpy(gpu_input, input, size, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(gpu_output, output, size, hipMemcpyHostToDevice));

	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

	hipEventRecord(beginKernel);
	shared_sobel_filter << <grid, block >> > (gpu_input, gpu_output, cols, rows, mask_dim);
	hipEventRecord(endKernel);

	CHECK_CUDA_ERROR(hipGetLastError());
	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	CHECK_CUDA_ERROR(hipMemcpy(output, gpu_output, size, hipMemcpyDeviceToHost));

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventSynchronize(endKernel);
	float elapsedKernel = 0.0f;
	float elapsedAll = 0.0f;

	hipEventElapsedTime(&elapsedKernel, beginKernel, endKernel);
	hipEventElapsedTime(&elapsedAll, start, stop);
	//printf("---- Sobel Filter ----\n");
	//printf("Total elapsed time in GPU (memory transfers are included) : %3.4f ms\n", elapsedAll);
	//printf("Elapsed time in GPU Kernel : %3.4f ms\n", elapsedKernel);
	//printf("---- ---------------------- ----\n");
	hipFree(gpu_input);
	hipFree(gpu_output);
	hipDeviceReset();
	return elapsedAll;
}
float sobel_filter_cpu(cv::Mat* inputImg, cv::Mat outputImg) {
	unsigned int offset = SOBEL_FILTER_SIZE / 2;
	const int rows = inputImg->rows;
	const int cols = inputImg->cols;

	int filter_x[3][3] = { {-1 , 0 , 1} , {-2 , 0 , 2 } , {-1 , 0 , 1} };
	int filter_y[3][3] = { {-1 , -2 , -1} , {0 , 0 , 0 } , {1 , 2 , 1} };

	auto begin = std::chrono::steady_clock::now();

	for (int i = 1; i < rows - 1; i++) {
		for (int j = 1; j < cols - 1; j++) {
			int gx = 0;
			int gy = 0;
			for (int m = 0; m < SOBEL_FILTER_SIZE; m++) {
				for (int n = 0; n < SOBEL_FILTER_SIZE; n++) {
					gx += inputImg->at<uchar>(i + m - offset, j + n - offset) * filter_x[m][n];
					gy += inputImg->at<uchar>(i + m - offset, j + n - offset) * filter_y[m][n];
				}
			}
			outputImg.at<uchar>(i, j) = static_cast<uchar>(sqrt(gx * gx + gy * gy));
		}
	}

	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin);
	return elapsed.count();
}
float sobel_filter_cpu_parallel(cv::Mat* inputImg, cv::Mat outputImg) {
	unsigned char* input = inputImg->data;
	unsigned char* output = outputImg.data;
	int cols = inputImg->cols;
	int rows = inputImg->rows;
	const unsigned short mask_dim = 3;

	int filter_x[3][3] = { {-1 , 0 , 1} , {-2 , 0 , 2 } , {-1 , 0 , 1} };
	int filter_y[3][3] = { {-1 , -2 , -1} , {0 , 0 , 0 } , {1 , 2 , 1} };

	std::vector <std::thread> threads;
	const int MAX_THREAD_SUPPORT = std::thread::hardware_concurrency();

	int stride = rows / MAX_THREAD_SUPPORT;

	auto begin = std::chrono::steady_clock::now();
	for (int i = 0; i < MAX_THREAD_SUPPORT; i++) {
		threads.push_back(std::thread([&, i]() {
			int range_start = stride * i;
			int range_end = (i == MAX_THREAD_SUPPORT - 1) ? cols : stride * (i + 1);

			for (int r = range_start; r < range_end; r++) { /*row loop*/
				for (int c = 0; c < cols; c++) { /*col loop*/
					if (r > 0 && r < rows - 1 && c > 0 && c < cols - 1) {
						int gx = 0;
						int gy = 0;
						for (int mr = 0; mr < mask_dim; mr++) { /*matrix row*/
							for (int mc = 0; mc < mask_dim; mc++) { /*matrix col*/
								int r_index = r + mr - 1;
								int c_index = c + mc - 1;
								gx += input[r_index * cols + c_index] * filter_x[mr][mc];
								gy += input[r_index * cols + c_index] * filter_y[mr][mc];
							}
						}
						output[r * cols + c] = static_cast<uchar>(sqrt(gx * gx + gy * gy));
					}
				}
			}
			}));
	}
	for (std::thread& th : threads) {
		th.join();
	}
	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - begin);
	return elapsed.count();
}