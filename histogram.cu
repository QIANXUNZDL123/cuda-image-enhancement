#include "hip/hip_runtime.h"
#include "dependencies.h"

__device__ int dev_histogram[256] = {0};
__device__ float dev_normalized_histogram[256] = {0};
__device__ float dev_cdf[256] = {0};
__device__ int dev_equalization_values[256] = {0};

/*color gpu variables*/

__device__ int dev_histogram_red[256] = {0};
__device__ float dev_normalized_histogram_red[256] = {0};
__device__ float dev_cdf_red[256] = {0};
__device__ int dev_equalization_values_red[256] = {0};

__device__ int dev_histogram_green[256] = {0};
__device__ float dev_normalized_histogram_green[256] = {0};
__device__ float dev_cdf_green[256] = {0};
__device__ int dev_equalization_values_green[256] = {0};

__device__ int dev_histogram_blue[256] = {0};
__device__ float dev_normalized_histogram_blue[256] = {0};
__device__ float dev_cdf_blue[256] = {0};
__device__ int dev_equalization_values_blue[256] = {0};

__global__ void k_1D_extract_histogram(unsigned char* input, int pixels) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);
	
	if (threadId >= pixels) {
		return;
	}
	atomicAdd(&dev_histogram[input[threadId]], 1);
}

__global__ void k_1D_extract_histogram_shared_mem(unsigned char* input, int pixels) { /*initialize histogram array*/
	__shared__ unsigned int cache[256];

	int threadIdInBlock = (threadIdx.x * blockDim.y) + threadIdx.y;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

	if (threadId >= pixels) {
		return;
	}

	if (threadIdInBlock < 256) {
		cache[threadIdInBlock] = 0;
	}
	__syncthreads();

	atomicAdd(&cache[(input[threadId])], 1);
	__syncthreads();

	if (threadIdInBlock < 256) {
		atomicAdd(&dev_histogram[threadIdInBlock], cache[threadIdInBlock]); /*there is no race condition in block threads. exits for threads in different blocks*/
	}
}

__global__ void k_1D_normalize_cdf_equalization(int pixels) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;

	dev_normalized_histogram[threadId] = dev_histogram[threadId] / (float)(pixels);
	__syncthreads();

	for (int i = 0; i <= threadId; i++) {
		sum += dev_normalized_histogram[i];
	}
	dev_cdf[threadId] = sum;
	dev_equalization_values[threadId] = int((dev_cdf[threadId] * 255.0f) + 0.5f);
}

__global__ void k_1D_normalize_cdf_equalization_shared_mem(int pixels) {
	__shared__ float cache_normalized_histogram[256];
	__shared__ float cache_cdf[256];

	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	cache_normalized_histogram[threadId] = dev_histogram[threadId] / (float)(pixels);
	__syncthreads();
	float sum = 0.0f;
	for (int i = 0; i <= threadId; i++) {
		sum += cache_normalized_histogram[i];
	}
	cache_cdf[threadId] = sum;
	dev_equalization_values[threadId] = int((cache_cdf[threadId] * 255.0f) + 0.5f);
}

__global__ void k_1D_histogram_equalization(unsigned char* input, int pixels) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

	if (threadId >= pixels) {
		return;
	}

	input[threadId] = static_cast<uchar>(dev_equalization_values[input[threadId]]);
}

__global__ void k_1D_histogram_equalization_shared_mem(unsigned char* input, int pixels) {
	__shared__ int cache_equalization_values[256];
	
	int thread_id_in_block = (threadIdx.x * blockDim.y) + threadIdx.y;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

	if (threadId >= pixels) {
		return;
	}
	if(thread_id_in_block < 256){
		cache_equalization_values[thread_id_in_block] = dev_equalization_values[thread_id_in_block];
	}
	__syncthreads();
	input[threadId] = static_cast<uchar>(cache_equalization_values[input[threadId]]);
}


__global__ void k_3D_extract_histogram(unsigned char* input, int rows, int cols) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x); 
	
	if (threadId >= rows * cols) {
		return;
	}
	switch (threadId % 3)
	{
	case 0:
		atomicAdd(&dev_histogram_red[input[threadId]], 1);
		break;
	case 1:
		atomicAdd(&dev_histogram_green[input[threadId]], 1);
		break;
	case 2:
		atomicAdd(&dev_histogram_blue[input[threadId]], 1);
		break;
	default:
		break;
	}
}

__global__ void k_3D_extract_histogram_shared_mem(unsigned char* input, int rows, int cols) {
	__shared__ unsigned int cache_histogram_red[256];
	__shared__ unsigned int cache_histogram_green[256];
	__shared__ unsigned int cache_histogram_blue[256];

	int threadIdInBlock = (threadIdx.x * blockDim.y) + threadIdx.y;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

	if (threadId >= rows * cols) {
		return;
	}

	if (threadIdInBlock < 256) {
		cache_histogram_red[threadIdInBlock] = 0;
		cache_histogram_green[threadIdInBlock] = 0;
		cache_histogram_blue[threadIdInBlock] = 0;
	}
	__syncthreads();

	switch (threadId % 3)
	{
	case 0:
		atomicAdd(&cache_histogram_red[(input[threadId])], 1);
		break;
	case 1:
		atomicAdd(&cache_histogram_green[(input[threadId])], 1);
		break;
	case 2:
		atomicAdd(&cache_histogram_blue[(input[threadId])], 1);
	default:
		break;
	}
	__syncthreads();

	if (threadIdInBlock < 256) {
		atomicAdd(&dev_histogram_red[threadIdInBlock], cache_histogram_red[threadIdInBlock]);
		atomicAdd(&dev_histogram_green[threadIdInBlock], cache_histogram_green[threadIdInBlock]);
		atomicAdd(&dev_histogram_blue[threadIdInBlock], cache_histogram_blue[threadIdInBlock]);
	}
}

__global__ void k_3D_normalize_cdf_equalization(int pixels) { /*1,256*/
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	dev_normalized_histogram_red[threadId] = dev_histogram_red[threadId] / (float)(pixels);
	dev_normalized_histogram_green[threadId] = dev_histogram_green[threadId] / (float)(pixels);
	dev_normalized_histogram_blue[threadId] = dev_histogram_blue[threadId] / (float)(pixels);
	__syncthreads();

	float sum_red = 0.0f, sum_green = 0.0f, sum_blue = 0.0f;
	for (int i = 0; i <= threadId; i++) {
		sum_red += dev_normalized_histogram_red[i];
		sum_green += dev_normalized_histogram_green[i];
		sum_blue += dev_normalized_histogram_blue[i];
	}
	dev_cdf_red[threadId] = sum_red;
	dev_cdf_green[threadId] = sum_green;
	dev_cdf_blue[threadId] = sum_blue;
	__syncthreads();

	dev_equalization_values_red[threadId] = int((dev_cdf_red[threadId] * 255.0f) + 0.5f);
	dev_equalization_values_green[threadId] = int((dev_cdf_green[threadId] * 255.0f) + 0.5f);
	dev_equalization_values_blue[threadId] = int((dev_cdf_blue[threadId] * 255.0f) + 0.5f);
}

__global__ void k_3D_normalize_cdf_equalization_shared_mem(int pixels) { /*1,256*/
	__shared__ float cache_normalized_histogram_red[256];
	__shared__ float cache_normalized_histogram_green[256];
	__shared__ float cache_normalized_histogram_blue[256];

	__shared__ float cache_cdf_red[256];
	__shared__ float cache_cdf_green[256];
	__shared__ float cache_cdf_blue[256];

	int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	cache_normalized_histogram_red[threadId] = dev_histogram_red[threadId] / (float)(pixels);
	cache_normalized_histogram_green[threadId] = dev_histogram_green[threadId] / (float)(pixels);
	cache_normalized_histogram_blue[threadId] = dev_histogram_blue[threadId] / (float)(pixels);
	__syncthreads();

	float sum_red = 0.0f, sum_green = 0.0f, sum_blue = 0.0f;
	for (int i = 0; i <= threadId; i++) {
		sum_red += cache_normalized_histogram_red[i];
		sum_green += cache_normalized_histogram_green[i];
		sum_blue += cache_normalized_histogram_blue[i];
	}
	cache_cdf_red[threadId] = sum_red;
	cache_cdf_green[threadId] = sum_green;
	cache_cdf_blue[threadId] = sum_blue;

	dev_equalization_values_red[threadId] = int((cache_cdf_red[threadId] * 255.0f) + 0.5f);
	dev_equalization_values_green[threadId] = int((cache_cdf_green[threadId] * 255.0f) + 0.5f);
	dev_equalization_values_blue[threadId] = int((cache_cdf_blue[threadId] * 255.0f) + 0.5f);
}

__global__ void k_3D_histogram_equalization(unsigned char* input, int rows , int cols) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

	if (threadId >= rows * cols) {
		return;
	}

	switch (threadId % 3)
	{
	case 0:
		input[threadId] = static_cast<uchar>(dev_equalization_values_red[input[threadId]]);
		break;
	case 1:
		input[threadId] = static_cast<uchar>(dev_equalization_values_green[input[threadId]]);
		break;
	case 2:
		input[threadId] = static_cast<uchar>(dev_equalization_values_blue[input[threadId]]);
		break;
	default:
		break;
	}
}

__global__ void k_3D_histogram_equalization_shared_mem(unsigned char* input, int rows, int cols) {
	__shared__ int cache_equalization_values_red[256];
	__shared__ int cache_equalization_values_green[256];
	__shared__ int cache_equalization_values_blue[256];
	
	int thread_id_in_block = (threadIdx.x * blockDim.y) + threadIdx.y;

	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int threadId = (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);

	if (threadId >= rows * cols) {
		return;
	}
	
	if (thread_id_in_block < 256) {
		cache_equalization_values_red[thread_id_in_block] = dev_equalization_values_red[thread_id_in_block];
		cache_equalization_values_green[thread_id_in_block] = dev_equalization_values_green[thread_id_in_block];
		cache_equalization_values_blue[thread_id_in_block] = dev_equalization_values_blue[thread_id_in_block];
	}
	__syncthreads();
	switch (threadId % 3)
	{
	case 0:
		input[threadId] = static_cast<uchar>(cache_equalization_values_red[input[threadId]]);
		break;
	case 1:
		input[threadId] = static_cast<uchar>(cache_equalization_values_green[input[threadId]]);
		break;
	case 2:
		input[threadId] = static_cast<uchar>(cache_equalization_values_blue[input[threadId]]);
		break;
	
	default:
		break;
	}
}

float histogram_equalization_gpu_3D(cv::Mat input_img, cv::Mat* output_img, bool sm) {
	unsigned char* gpu_input = nullptr;

	unsigned char* input = input_img.data;
	unsigned char* output = output_img->data;

	unsigned int cols = input_img.cols * 3;
	unsigned int rows = input_img.rows;
	unsigned int pixels = input_img.cols * input_img.rows;
	unsigned long int size = rows * cols * sizeof(unsigned char);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	CHECK_CUDA_ERROR(hipMalloc((unsigned char**)&gpu_input, size));
	CHECK_CUDA_ERROR(hipMemcpy(gpu_input, input, size, hipMemcpyHostToDevice));

	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

	if(sm){
		k_3D_extract_histogram_shared_mem << <grid, block >> > (gpu_input, rows, cols);
		CHECK_CUDA_ERROR(hipDeviceSynchronize());

		k_3D_normalize_cdf_equalization_shared_mem << <1, 256 >> > (pixels);
		CHECK_CUDA_ERROR(hipDeviceSynchronize());

		k_3D_histogram_equalization_shared_mem << <grid, block >> > (gpu_input, rows, cols);
	}else{
		k_3D_extract_histogram << <grid, block >> > (gpu_input, rows, cols);
		CHECK_CUDA_ERROR(hipDeviceSynchronize());

		k_3D_normalize_cdf_equalization<< <1, 256 >> > (pixels);
		CHECK_CUDA_ERROR(hipDeviceSynchronize());

		k_3D_histogram_equalization<< <grid, block >> > (gpu_input, rows, cols);
	}
	
	CHECK_CUDA_ERROR(hipMemcpy(output, gpu_input, size, hipMemcpyDeviceToHost));

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float gpuElapsedTime = 0;
	hipEventElapsedTime(&gpuElapsedTime, start, stop);

	hipFree(gpu_input);
	hipDeviceReset();
	return gpuElapsedTime;
}

float histogram_equalization_gpu_1D(cv::Mat inputImg, cv::Mat* outputImg, bool sm) {
	unsigned char* gpu_input = nullptr;

	unsigned char* input = inputImg.data;
	unsigned char* output = outputImg->data;

	unsigned int cols = inputImg.cols;
	unsigned int rows = inputImg.rows;
	unsigned int pixels = cols * rows;
	unsigned long int size = pixels * sizeof(unsigned char); /*Single channel image.*/

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	CHECK_CUDA_ERROR(hipMalloc((unsigned char**)&gpu_input, size));
	CHECK_CUDA_ERROR(hipMemcpy(gpu_input, input, size, hipMemcpyHostToDevice));

	hipStreamSynchronize(0);

	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

	if(sm){
		k_1D_extract_histogram_shared_mem << <grid, block >> > (gpu_input, pixels);
		CHECK_CUDA_ERROR(hipDeviceSynchronize());

		k_1D_normalize_cdf_equalization_shared_mem << <1, 256 >> > (pixels);
		CHECK_CUDA_ERROR(hipDeviceSynchronize());

		k_1D_histogram_equalization_shared_mem << <grid, block >> > (gpu_input, pixels);
	}else{
		k_1D_extract_histogram << <grid, block >> > (gpu_input, pixels);
		CHECK_CUDA_ERROR(hipDeviceSynchronize());

		k_1D_normalize_cdf_equalization << <1, 256 >> > (pixels);
		CHECK_CUDA_ERROR(hipDeviceSynchronize());

		k_1D_histogram_equalization<< <grid, block >> > (gpu_input, pixels);
	}
	
	CHECK_CUDA_ERROR(hipMemcpy(output, gpu_input, size, hipMemcpyDeviceToHost));

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float gpuElapsedTime = 0;
	hipEventElapsedTime(&gpuElapsedTime, start, stop);

	hipFree(gpu_input);
	hipDeviceReset();
	return gpuElapsedTime;
}
float histogram_equalization_cpu_1D(cv::Mat inputImg, cv::Mat* outputImg) {
	unsigned char* input = inputImg.data;
	unsigned char* output = outputImg->data;

	int histogram[256] = { 0 };
	float cdf[256] = { 0 };
	float normalizedHistogram[256] = { 0 };
	int equalization[256] = { 0 };
	
	int pixels = inputImg.cols * inputImg.rows;

	auto start = std::chrono::steady_clock::now();

	for (int i = 0; i < pixels; i++) {
		histogram[input[i]]++;
	}

	for (int i = 0; i < 256; i++) { 
		normalizedHistogram[i] = (histogram[i] / (float)pixels);
	}

	cdf[0] = normalizedHistogram[0];
	for (int i = 1; i < 256; i++) {
		cdf[i] = cdf[i - 1] + normalizedHistogram[i];
	}

	for (int i = 0; i < 256; i++) {
		equalization[i] = int((cdf[i] * 255.0f) + 0.5f);
	}

	for (int i = 0; i < pixels; i++) {
		output[i] = equalization[input[i]];
	}

	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start) / 1000.0f;
	return elapsed.count();
}

float histogram_equalization_cpu_3D(cv::Mat input_img, cv::Mat* output_img) {
	unsigned char* input = input_img.data;
	unsigned char* output = output_img->data;

	int histogram_red[256] = { 0 };
	int histogram_green[256] = { 0 };
	int histogram_blue[256] = { 0 };

	float normalize_histogram_red[256] = { 0 };
	float normalize_histogram_green[256] = { 0 };
	float normalize_histogram_blue[256] = { 0 };

	float cdf_red[256] = { 0 };
	float cdf_green[256] = { 0 };
	float cdf_blue[256] = { 0 };

	int equalization_red[256] = { 0 };
	int equalization_green[256] = { 0 };
	int equalization_blue[256] = { 0 };

	int pixels = input_img.cols * input_img.rows;
	int size = pixels * 3 * sizeof(unsigned char);

	auto start = std::chrono::steady_clock::now();

	for (int i = 0; i < pixels; i++) { /*Calculating histogram of input image*/
		histogram_red[input[i * 3]]++;
		histogram_green[input[i * 3 + 1]]++;
		histogram_blue[input[i * 3 + 2]]++;
	}
	for (int i = 0; i < 256; i++) { /*Calculating normalized histogram (better calculation speed)*/
		normalize_histogram_red[i] = (histogram_red[i] / (float)pixels);
		normalize_histogram_green[i] = (histogram_green[i] / (float)pixels);
		normalize_histogram_blue[i] = (histogram_blue[i] / (float)pixels);
	}

	cdf_red[0] = normalize_histogram_red[0];
	cdf_green[0] = normalize_histogram_green[0];
	cdf_blue[0] = normalize_histogram_blue[0];

	for (int i = 1; i < 256; i++) { /*Generating CDF array*/
		cdf_red[i] = cdf_red[i - 1] + normalize_histogram_red[i];
		cdf_green[i] = cdf_green[i - 1] + normalize_histogram_green[i];
		cdf_blue[i] = cdf_blue[i - 1] + normalize_histogram_blue[i];
	}

	for (int i = 0; i < 256; i++) { /*Generating new pixel intensity values then assign them*/
		equalization_red[i] = int((cdf_red[i] * 255.0f) + 0.5f);
		equalization_green[i] = int((cdf_green[i] * 255.0f) + 0.5f);
		equalization_blue[i] = int((cdf_blue[i] * 255.0f) + 0.5f);
	}
	for (int i = 0; i < pixels; i++) {
		output[i * 3] = equalization_red[input[i * 3]];
		output[i * 3 + 1] = equalization_green[input[i * 3 + 1]];
		output[i * 3 + 2] = equalization_blue[input[i * 3 + 2]];
	}

	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start) / 1000.0f;
	return elapsed.count();
}

float histogram_equalization_cpu_parallel_1D(cv::Mat inputImg, cv::Mat* outputImg) {
	const unsigned char* input = inputImg.data;
	unsigned char* output = outputImg->data;

	const unsigned int rows = inputImg.rows;
	const unsigned int cols = inputImg.cols;

	int histogram[256] = { 0 };
	float normalizedHistogram[256] = { 0 };
	float cdf[256] = { 0 };
	int equalization[256] = { 0 };
	int pixels = cols * rows;

	std::vector <std::thread> threads;
	std::mutex mtx;
	std::condition_variable cv;

	const int MAX_THREAD_SUPPORT = 12;
	const int stride = rows / MAX_THREAD_SUPPORT;
	const int stride_for_256 = 256 / MAX_THREAD_SUPPORT;

	int step1_count = 0;
	int step2_count = 0;
	int step3_count = 0;
	int step4_count = 0;
	auto start = std::chrono::steady_clock::now();

	for (int id = 0; id < MAX_THREAD_SUPPORT; id++) {
		threads.push_back(std::thread([&,id] () {
			int range_start = stride * id;
			int range_end = (id == MAX_THREAD_SUPPORT - 1) ? rows : stride * (id + 1);

			int t_histogram[256] = {0};

			for (int r = range_start; r < range_end; r++) {
				for (int c = 0; c < cols; c++) {
					{
						t_histogram[input[r * cols + c]]++;
					}
				}
			}
			
			{
				std::unique_lock<std::mutex> lck(mtx);
				for(int i = 0 ; i < 256 ; i++){
					histogram[i] += t_histogram[i];
				}
			}
			
			{
				std::unique_lock<std::mutex> lck(mtx);
				if (++step1_count == MAX_THREAD_SUPPORT) {
					cv.notify_all();
				}
				else {
					cv.wait(lck);
				}
			}

			range_start = stride_for_256 * id;
			range_end = (id == MAX_THREAD_SUPPORT - 1) ? 256 : stride_for_256 * (id + 1);

			for (int i = range_start; i < range_end; i++) {
				normalizedHistogram[i] = histogram[i] / (float)pixels;
			}
			{
				std::unique_lock<std::mutex> lck(mtx);
				if (++step2_count == MAX_THREAD_SUPPORT) {
					cv.notify_all();
				}
				else {
					cv.wait(lck);
				}
			}
			cdf[0] = normalizedHistogram[0];

			for (int i = range_start; i < range_end; i++) {
				if(i == 0)
					continue;
				float sum = 0.0f;
				for (int j = 0; j <= i; j++) {
					sum += normalizedHistogram[j];
				}
				cdf[i] = sum;
			}
			{
				std::unique_lock<std::mutex> lck(mtx);
				if (++step3_count == MAX_THREAD_SUPPORT) {
					cv.notify_all();
				}
				else {
					cv.wait(lck);
				}
			}

			for (int i = range_start; i < range_end; i++) {
				equalization[i] = int((cdf[i] * 255.0f) + 0.5f);
			}

			{
				std::unique_lock<std::mutex> lck(mtx);
				if (++step4_count == MAX_THREAD_SUPPORT) {
					cv.notify_all();
				}
				else {
					cv.wait(lck);
				}
			}

			range_start = stride * id;
			range_end = (id == MAX_THREAD_SUPPORT - 1) ? rows : stride * (id + 1);
			for (int r = range_start; r < range_end; r++) {
				for (int c = 0; c < cols; c++) {
					int index = r * cols + c;
					output[index] = equalization[input[index]];
				}
			}
			}));
	}
	for (std::thread& thread : threads) {
		thread.join();
	}
	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start) / 1000.0f;
	return elapsed.count();
}

float histogram_equalization_cpu_parallel_3D(cv::Mat inputImg, cv::Mat* outputImg) {
	const unsigned char* input = inputImg.data;
	unsigned char* output = outputImg->data;

	const unsigned int rows = inputImg.rows;
	const unsigned int cols = inputImg.cols;

	int histogram_red[256] = { 0 };
	int histogram_green[256] = { 0 };
	int histogram_blue[256] = { 0 };

	float normalize_histogram_red[256] = { 0 };
	float normalize_histogram_green[256] = { 0 };
	float normalize_histogram_blue[256] = { 0 };

	float cdf_red[256] = { 0 };
	float cdf_green[256] = { 0 };
	float cdf_blue[256] = { 0 };

	int equalization_red[256] = { 0 };
	int equalization_green[256] = { 0 };
	int equalization_blue[256] = { 0 };
	int pixels = cols * rows;

	std::vector <std::thread> threads;
	std::mutex mtx;
	std::condition_variable cv;

	const int MAX_THREAD_SUPPORT = 12;
	const int stride = rows / MAX_THREAD_SUPPORT;
	const int stride_for_256 = 256 / MAX_THREAD_SUPPORT;

	int step1_count = 0;
	int step2_count = 0;
	int step3_count = 0;
	int step4_count = 0;
	auto start = std::chrono::steady_clock::now();

	for (int id = 0; id < MAX_THREAD_SUPPORT; id++) {
		threads.push_back(std::thread([&,id] () {
			int range_start = stride * id;
			int range_end = (id == MAX_THREAD_SUPPORT - 1) ? rows : stride * (id + 1);

			int local_histogram_red[256] = {0};
			int local_histogram_green[256] = {0};
			int local_histogram_blue[256] = {0};

			for (int r = range_start; r < range_end; r++) {
				for (int c = 0; c < cols; c++) {
					{
						int index = (r * cols + c) * 3;
						local_histogram_red[input[index]]++;
						local_histogram_green[input[index + 1]]++;
						local_histogram_blue[input[index + 2]]++;
					}
				}
			}
			{
				std::unique_lock<std::mutex> lck(mtx);
				for(int i = 0 ; i < 256 ; i++){
					histogram_red[i] += local_histogram_red[i];
					histogram_green[i] += local_histogram_green[i];
					histogram_blue[i] += local_histogram_blue[i];
				}
			}
			
			{
				std::unique_lock<std::mutex> lck(mtx);
				if (++step1_count == MAX_THREAD_SUPPORT) {
					cv.notify_all();
				}
				else {
					cv.wait(lck);
				}
			}

			range_start = stride_for_256 * id;
			range_end = (id == MAX_THREAD_SUPPORT - 1) ? 256 : stride_for_256 * (id + 1);

			for (int i = range_start; i < range_end; i++) {
				normalize_histogram_red[i] = histogram_red[i] / (float)pixels;
				normalize_histogram_green[i] = histogram_green[i] / (float)pixels;
				normalize_histogram_blue[i] = histogram_blue[i] / (float)pixels;
			}
			{
				std::unique_lock<std::mutex> lck(mtx);
				if (++step2_count == MAX_THREAD_SUPPORT) {
					cv.notify_all();
				}
				else {
					cv.wait(lck);
				}
			}
			cdf_red[0] = normalize_histogram_red[0];
			cdf_green[0] = normalize_histogram_green[0];
			cdf_blue[0] = normalize_histogram_blue[0];

			for (int i = range_start; i < range_end; i++) {
				float sum_red = 0;
				float sum_green = 0;
				float sum_blue = 0;
				for (int j = 0; j <= i; j++) {
					sum_red += normalize_histogram_red[j];
					sum_green += normalize_histogram_green[j];
					sum_blue += normalize_histogram_blue[j];
				}
				cdf_red[i] = sum_red;
				cdf_green[i] = sum_green;
				cdf_blue[i] = sum_blue;
			}
			{
				std::unique_lock<std::mutex> lck(mtx);
				if (++step3_count == MAX_THREAD_SUPPORT) {
					cv.notify_all();
				}
				else {
					cv.wait(lck);
				}
			}

			for (int i = range_start; i < range_end; i++) {
				equalization_red[i] = int((cdf_red[i] * 255.0f) + 0.5f);
				equalization_green[i] = int((cdf_green[i] * 255.0f) + 0.5f);
				equalization_blue[i] = int((cdf_blue[i] * 255.0f) + 0.5f);
			}

			{
				std::unique_lock<std::mutex> lck(mtx);
				if (++step4_count == MAX_THREAD_SUPPORT) {
					cv.notify_all();
				}
				else {
					cv.wait(lck);
				}
			}

			range_start = stride * id;
			range_end = (id == MAX_THREAD_SUPPORT - 1) ? rows : stride * (id + 1);
			for (int r = range_start; r < range_end; r++) {
				for (int c = 0; c < cols; c++) {
					int index = (r * cols + c) * 3;
					output[index] = equalization_red[input[index]];
					output[index + 1] = equalization_green[input[index + 1]];
					output[index + 2] = equalization_blue[input[index + 2]];
				}
			}
			}));
	}
	for (std::thread& thread : threads) {
		thread.join();
	}
	auto end = std::chrono::steady_clock::now();
	auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start) / 1000.0f;
	return elapsed.count();
}